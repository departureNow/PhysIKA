#include "hip/hip_runtime.h"
#include "SortCuda.h"
#include <cassert>
#include <cfloat>
#include "hip/hip_runtime.h"
#include ""
#include "hip/device_functions.h"
#include <climits>
#include<cstdio>
#include<cstdlib>
#include<string>



namespace PhysIKA {
#define MAX_LENGTH 1024
	template<typename T>
	SortCuda<T>::SortCuda() {
		this->sortLength = 0;
	}

	template<typename T>
	SortCuda<T>::SortCuda(int length) {
		this->sortLength = length;
	}

	template<typename T>
	SortCuda<T>::~SortCuda() {
		free(this->host_input);
		free(this->host_res);
		hipFree(this->device_input);
		hipFree(this->device_res);
	}
/*
	__device__ int getBinaryByN(int num, int id) {
		int count = 0;
		int temp;
		while (num != 0) {
			temp = num % 2;
			num = num / 2;
			count++;
			if (count == id) {
				return temp;
			}
		}
		return 0;
	}

	__global__ void deviceRadixSort(int *arr, int length) {
		__shared__ int a0[MAX_LENGTH];
		__shared__ int a1[MAX_LENGTH];
		int id = threadIdx.x;
		int k0 = 0;
		int k1 = 0;
		for (int i = 0; i < length; i++) {
			int x = getBinaryByN(arr[i], id + 1);
			if (x == 0) {
				a0[k0] = arr[i];
				k0++;
			}
			else if (x == 1) {
				a1[k1] = arr[i];
				k1++;
			}
			__syncthreads();
		}
		for (int i = 0; i < k0; i++) {
			arr[i] = a0[i];
		}
		__syncthreads();
		for (int i = k0, i1 = 0; i < k0 + k1; i1++, i++) {
			arr[i] = a1[i1];
		}
		__syncthreads();
	}
	void SortCuda::radixSort(int * arr, int length) {
		numMalloc(arr, length);
		int maxNum = INT_MIN;
		for (int i = 0; i < length; i++) {
			if (arr[i] > maxNum) {
				maxNum = arr[i];
			}
		}
		int bitLength = int2bit(maxNum);
		deviceRadixSort << <1, bitLength>> > (num, length);
		hipMemcpy((void*)arr, (void*)num, length * sizeof(int), hipMemcpyDeviceToHost);
		hipFree(num);
	}
	*/

	template<typename T>
	__device__ inline void swap(T &num1, T &num2, int dir) {
		T temp;
		if ((num1 > num2) == dir) {
			temp = num1;
			num1 = num2;
			num2 = temp;
		}
	}

	template<typename T>
	__global__ void smallBinoticSort(T *arr, T *res, int length, int dir) {
		__shared__ T buf[MAX_LENGTH];
		buf[threadIdx.x] = arr[threadIdx.x];
		buf[threadIdx.x + (MAX_LENGTH / 2)] = arr[threadIdx.x + (MAX_LENGTH / 2)];
		//printf("%d\n", arr[threadIdx.x]);
		//printf("%d\n", arr[threadIdx.x+ (MAX_LENGTH / 2)]);
		__syncthreads();
		for (int size = 2; size < length; size<<=1) {
			int d = dir ^ ((threadIdx.x & (size / 2)) != 0);
			for (int j = size /2; j > 0; j >>= 1) {
				__syncthreads();
				int pos = 2 * threadIdx.x - (threadIdx.x&(j - 1));
				swap(buf[pos], buf[pos + j], d);
			}
		}
		for (int j = length/2; j > 0; j >>= 1) {
			__syncthreads();
			int pos = 2 * threadIdx.x - (threadIdx.x&(j - 1));
			swap(buf[pos], buf[pos + j], dir);
		}
		__syncthreads();
		res[threadIdx.x] = buf[threadIdx.x];
		res[threadIdx.x + MAX_LENGTH / 2] = buf[threadIdx.x + MAX_LENGTH / 2];
	}

	template<typename T>
	__global__ void firstBinoticSort(T *arr, T *res) {
		__shared__ T buf[MAX_LENGTH];
		int id = blockIdx.x * MAX_LENGTH + threadIdx.x;
		buf[threadIdx.x] = arr[id];
		buf[threadIdx.x + (MAX_LENGTH / 2)] = arr[id + (MAX_LENGTH / 2)];
		//printf("%d\n", arr[threadIdx.x]);
		//printf("%d\n", arr[threadIdx.x+ (MAX_LENGTH / 2)]);
		__syncthreads();
		for (int size = 2; size < MAX_LENGTH; size <<= 1) {
			int d = (threadIdx.x & (size / 2)) != 0;
			for (int j = size / 2; j > 0; j >>= 1) {
				__syncthreads();
				int pos = 2 * threadIdx.x - (threadIdx.x&(j - 1));
				swap(buf[pos], buf[pos + j], d);
			}
		}
	//printf("%d\n", blockIdx.x);
		int d = blockIdx.x & 1; //��ż
		for (int j = MAX_LENGTH / 2; j > 0; j >>= 1) {
			__syncthreads();
			int pos = 2 * threadIdx.x - (threadIdx.x&(j - 1));
			swap(buf[pos], buf[pos + j], d);
		}
		__syncthreads();
		res[id] = buf[threadIdx.x];
		res[id + MAX_LENGTH / 2] = buf[threadIdx.x + MAX_LENGTH / 2];
		//printf("%d\n", res[threadIdx.x]);
	}

	template<typename T>
	__global__ void bitonicMergeLarge(T *arr, T *res, int length,int size,int j, int dir) {
		int id = blockIdx.x*blockDim.x + threadIdx.x;
		int com = id & (length/2 - 1);

		int d = dir^((com&(size / 2)) != 0);
		int pos = 2 * id - (id&(j - 1));

		int num1 = arr[pos];
		int num2 = arr[pos + j];
		swap(num1, num2, d);

		res[pos] = num1;
		res[pos + j] = num2;
	}

	template<typename T>
	__global__ void bitonicMergeSmall(T *arr, T *res, int length, int size, int dir) {
		__shared__ T buf[MAX_LENGTH];
		int id = blockIdx.x*MAX_LENGTH + threadIdx.x;
		buf[threadIdx.x] = arr[id];
		buf[threadIdx.x + MAX_LENGTH / 2] = arr[id + MAX_LENGTH / 2];

		int id1 = blockIdx.x * blockDim.x + threadIdx.x;
		int com = id1 & ((length / 2) - 1);
		int d = dir ^ ((com&(size / 2)) != 0);

		for (int j = MAX_LENGTH / 2; j > 0; j >>= 1) {
			__syncthreads();
			int pos = 2 * threadIdx.x - (threadIdx.x&(j - 1));
			swap(buf[pos], buf[pos + j], d);

		}
		__syncthreads();

		res[id] = buf[threadIdx.x];
		res[id + MAX_LENGTH / 2] = buf[threadIdx.x + MAX_LENGTH / 2];
	}

	/*
	arr������������
	length�����鳤��
	dir�����Ƶ������ߵݼ���1Ϊ������0Ϊ�ݼ�
	*/

	template<typename T>
	void SortCuda<T>::binoticSort(T * arr, int length, int dir){

		if (length <= MAX_LENGTH) {
			dataMalloc(MAX_LENGTH);

			arrExpand(arr, host_input, length, MAX_LENGTH);

			hipMemcpy(device_input, host_input, MAX_LENGTH * sizeof(T), hipMemcpyHostToDevice);
			hipDeviceSynchronize();
			smallBinoticSort<T> << <1, MAX_LENGTH/2 >> > (device_input, device_res, MAX_LENGTH, dir);
			hipDeviceSynchronize();
			hipMemcpy(host_res, device_res, MAX_LENGTH * sizeof(T), hipMemcpyDeviceToHost);

			if (dir == 1) {
				for (int i = 0; i < length; i++) {
					arr[i] = host_res[i];
				}
			}else if (dir == 0) {
				for (int i = MAX_LENGTH-length , j=0; i <MAX_LENGTH; i++,j++) {
					arr[j] = host_res[i];
				}
			}
			/*for (uint i = 0; i < length; i++) {
				printf("%d\n", arr[i]);
			}
			printf("\n");*/

		}
		else {
			if (length&(length-1)==0) {
				dataMalloc(length);
				for (int i = 0; i < length; i++) {
					host_input[i] = arr[i];
				}
				hipMemcpy(device_input, host_input, length * sizeof(T), hipMemcpyHostToDevice);
				hipDeviceSynchronize();
				int blockNum = length / MAX_LENGTH;
				int threadNum = MAX_LENGTH / 2;
				//printf("%d\n\n", length);
				firstBinoticSort<T> << <blockNum, threadNum >> > (device_input, device_res);

				for (int size = 2 * MAX_LENGTH; size <= length; size <<= 1) {
					for (int j = size / 2; j > 0; j >>= 1) {
						if (j >= MAX_LENGTH) {
							bitonicMergeLarge<T> << <length / MAX_LENGTH, MAX_LENGTH/2 >> > (device_res, device_res, length, size, j, dir);
						}
						else {
							bitonicMergeSmall<T> << <blockNum, threadNum >> > (device_res, device_res, length, size, dir);
							break;
						}
					}
				}
				hipDeviceSynchronize();
				hipMemcpy(host_res, device_res, length * sizeof(T), hipMemcpyDeviceToHost);

				for (int i = 0; i < length; i++) {
					arr[i] = host_res[i];
				}
			}else{
				int temp = log(length) / log(2);
				int new_len = pow(2,temp+1);
				//printf("%d\n", new_len);
				dataMalloc(new_len);
				arrExpand(arr, this->host_input, length, new_len);
				hipMemcpy(device_input, host_input, new_len * sizeof(T), hipMemcpyHostToDevice);
				hipDeviceSynchronize();
				int blockNum = new_len / MAX_LENGTH;
				int threadNum = MAX_LENGTH / 2;

				firstBinoticSort<T> << <blockNum, threadNum >> > (device_input, device_res);

				for (int size = 2 * MAX_LENGTH; size <= new_len; size <<= 1) {
					for (int j = size / 2; j > 0; j >>= 1) {
						if (j >= MAX_LENGTH) {
							bitonicMergeLarge<T> << <new_len / MAX_LENGTH, MAX_LENGTH/2 >> > (device_res, device_res, new_len, size, j, dir);
						}
						else {
							bitonicMergeSmall<T> << <blockNum, threadNum >> > (device_res, device_res, new_len, size, dir);
							break;
						}
					}
				}
				hipDeviceSynchronize();
				hipMemcpy(host_res, device_res, new_len * sizeof(T), hipMemcpyDeviceToHost);
				if (dir == 1) {
					for (int i = 0; i < length; i++) {
						arr[i] = host_res[i];
					}
				}
				else if (dir == 0) {
					for (int i = new_len - length, j = 0; i < new_len; i++, j++) {
						arr[j] = host_res[i];
					}
				}
			}

		}


	}

	template<typename T>
	void SortCuda<T>::dataMalloc(T length)
	{
		if (this->device_input == nullptr) {
			hipFree(this->device_input);
		}
		if (this->device_res == nullptr) {
			hipFree(this->device_res);
		}
		if (this->host_input == nullptr) {
			free(this->host_input);
		}
		if (this->host_res == nullptr) {
			free(this->host_res);
		}
		this->host_res = (T*)malloc(sizeof(T)*length);
		this->host_input = (T*)malloc(sizeof(T)*length);
		hipMalloc((void **)&this->device_input, length * sizeof(T));
		hipMalloc((void **)&this->device_res, length * sizeof(T));
	}

	template<typename T>
	void SortCuda<T>::arrExpand(T * arr, T * arr_new, int length, int new_length)
	{
		int com1 = 1;
		float com2 = 0;
		double com3 = 0.5;
		for (int i = 0; i < length; i++) {
			arr_new[i] = arr[i];
		}
		//const type_info &nInfo = typeid(T);
		//char* name = ;
		//printf("%s\n", nInfo.name());
		if (typeid(T)==typeid(com1) ){
			printf("-----------------------------------\n");
			for (int i = length; i < new_length; i++) {
				arr_new[i] = INT_MAX;
			}
		}
		else if (typeid(T) == typeid(com2)) {
			for (int i = length; i < new_length; i++) {
				arr_new[i] = FLT_MAX;
			}
		}
		else if (typeid(T) == typeid(com3)) {
			for (int i = length; i < new_length; i++) {
				arr_new[i] = DBL_MAX;
			}
		}
		/*switch (name) {
			case "int":		
				for (int i = length; i < new_length; i++) {
					arr_new[i] = INT_MAX;
				}
			case "float":
				for (int i = length; i < new_length; i++) {
					arr_new[i] = FLT_MAX;
				}
			case "double":
				for (int i = length; i < new_length; i++) {
					arr_new[i] = DBL_MAX;
				}
		}*/

	}

	
	/*
	int SortCuda::int2bit(int n) {
		int count = 0;
		while (n != 0) {
			n = n / 2;
			count++;
		}
		return count;
	}



	void SortCuda::numMalloc(int *arr, int length) {
		hipMalloc((void**)&num, length * sizeof(int));
		hipMemcpy((void*)num, (void*)arr, length * sizeof(int), hipMemcpyHostToDevice);
	}*/
}