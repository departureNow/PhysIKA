#include "hip/hip_runtime.h"
#include "Sort.h"
#include <cassert>
#include <cfloat>
#include <hip/hip_runtime.h>
#include "cuda_utilities.h"
#include "sharedmem.h"
#include "Functional.h"


namespace PhysIKA {
#define MAX_LENGTH 10000
	Sort::Sort() {

	}

	Sort::Sort(int length) {

	}

	Sort::~Sort() {
		hipFree(num);
	}
	int * Sort::radixSort(int * arr, int length) {
		numMalloc(arr, length);
		int maxNum = INT_MIN;
		for (int i = 0; i < length; i++) {
			if (arr[i] > maxNum) {
				maxNum = arr[i];
			}
		}
		int bitLength = int2bit(maxNum);
		deviceRadixSort << <1, bitLength>> > (arr, length);
		hipMemcpy((void*)arr, (void*)num, length * sizeof(int), hipMemcpyDeviceToHost);
		return arr;
	}

	int Sort::int2bit(int n) {
		int count = 0;
		while (n != 0) {
			n = n / 2;
			count++;
		}
		return count;
	}

	__device__ int getBinaryByN(int num, int id) {
		int count = 0;
		int temp;
		while (num != 0) {
			temp = num % 2;
			num = num / 2;
			count++;
			if (count == id) {
				return temp;
			}
		}
		return 0;
	}

	__global__ void deviceRadixSort(int *arr, int length) {
		extern __shared__ int a0[MAX_LENGTH];
		extern __shared__ int a1[MAX_LENGTH];
		int id = threadIdx.x;
		int k0 = 0;
		int k1 = 0;
		for (int i = 0; i < length; i++) {
			int x = getBinaryByN(arr[i], id + 1);
			if (x == 0) {
				a0[k0] = arr[i];
				k0++;
			}else if (x == 1) {
				a1[k1] = arr[i];
				k1++;
			}
			__syncthreads();
		}
		for (int i = 0; i < k0; i++) {
			arr[i] = a0[i];
		}
		__syncthreads();
		for (int i = k0, i1 = 0; i < k0 + k1; i1++, i++) {
			arr[i] = a1[i1];
		}
		__syncthreads();
	}


	void Sort::numMalloc(int *arr, int length) {
		hipMalloc((void**)&num, length * sizeof(int));
		hipMemcpy((void*)num, (void*)arr, length * sizeof(int), hipMemcpyHostToDevice);
	}
}